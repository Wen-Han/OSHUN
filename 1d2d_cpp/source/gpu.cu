#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "gpu.h"

#define BLOCKSIZE_x 32
#define BLOCKSIZE_y 4

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %d, %s %s %d\n", code, hipGetErrorString(code), file, line);
      if (abort) { exit(code); }
   }
}

extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

        case HIPSPARSE_STATUS_SUCCESS:
            return "HIPSPARSE_STATUS_SUCCESS";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "HIPSPARSE_STATUS_NOT_INITIALIZED";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "HIPSPARSE_STATUS_ALLOC_FAILED";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "HIPSPARSE_STATUS_INVALID_VALUE";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "HIPSPARSE_STATUS_ARCH_MISMATCH";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "HIPSPARSE_STATUS_MAPPING_ERROR";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "HIPSPARSE_STATUS_EXECUTION_FAILED";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "HIPSPARSE_STATUS_INTERNAL_ERROR";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

        case HIPSPARSE_STATUS_ZERO_PIVOT:
            return "HIPSPARSE_STATUS_ZERO_PIVOT";
    }

    return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if(HIPSPARSE_STATUS_SUCCESS != err) {
        fprintf(stderr, "CUSPARSE error in file '%s', line %Ndims \n objs %s\nerror %Ndims: %s\nterminating! \n objs",__FILE__, __LINE__,err, \
                                _cusparseGetErrorEnum(err)); \
        hipDeviceReset(); assert(0); \
    }
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/

void GPU_interface_routines::AllocateMatrixSystemOnHost(int totalsize, 
                double& ld, double &dd, double &ud, double &fin)
{
    double a;

    hipHostAlloc((void**)ld, totalsize*sizeof(a),hipHostMallocDefault);
    hipHostAlloc((void**)dd, totalsize*sizeof(a),hipHostMallocDefault);
    hipHostAlloc((void**)ud, totalsize*sizeof(a),hipHostMallocDefault);
    hipHostAlloc((void**)fin, totalsize*sizeof(a),hipHostMallocDefault);

}
void GPU_interface_routines::TDsolve( int calculations_per_loop, int n_systems,
                            double *ld, double *dd, double *ud, double *fin, 
                            int device)
{
    hipSetDevice(device);
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

    const int N     =  n_systems*calculations_per_loop;        // --- Size of the linear system

    // std::cout << "\n 10 \n";
    // // --- Lower diagonal, diagonal and upper diagonal of the system matrix
    double *d_ld;   gpuErrchk(hipMalloc(&d_ld, N * sizeof(double)));
    double *d_d;    gpuErrchk(hipMalloc(&d_d,  N * sizeof(double)));
    double *d_ud;   gpuErrchk(hipMalloc(&d_ud, N * sizeof(double)));

    // double *d_ld;   double *d_d;  double *d_ud; double *d_x;
    // hipGetSymbolAddress((void **)&d_ld, lowerdiagonal);
    // hipGetSymbolAddress((void **)&d_d, diagonal);
    // hipGetSymbolAddress((void **)&d_ud, upperdiagonal);
    // hipGetSymbolAddress((void **)&d_x, solution);

    gpuErrchk(hipMemcpy(d_ld, ld, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 11 \n";
    gpuErrchk(hipMemcpy(d_d,  dd, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 12 \n";
    gpuErrchk(hipMemcpy(d_ud, ud, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 13 \n";

    // --- Allocating and defining dense device data vectors
    double *d_x;        gpuErrchk(hipMalloc(&d_x, N * sizeof(double)));   
    gpuErrchk(hipMemcpy(d_x, fin, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 14 \n";

    // --- Solve for solution
    cusparseSafeCall(cusparseDgtsvStridedBatch(handle, calculations_per_loop, d_ld, d_d, d_ud, d_x, n_systems, calculations_per_loop));

    // --- Copy back into host
    hipMemcpy(fin, d_x, N * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_ld);hipFree(d_ud);hipFree(d_d);hipFree(d_x);
    cusparseSafeCall(hipsparseDestroy(handle));
}
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
FokkerPlanckOnGPU::FokkerPlanckOnGPU()
{
    calc_per_loop = 0;
    n_sys = 0;

}
void FokkerPlanckOnGPU::initialize(int calculations_per_loop, int n_systems, int device)
        // : calc_per_loop(calculations_per_loop), n_sys(n_systems)
{
    calc_per_loop = calculations_per_loop;
    n_sys = n_systems;

    hipSetDevice(device);
    gpuErrchk(hipMalloc(&d_ld, calc_per_loop * n_sys * sizeof(double)));
    gpuErrchk(hipMalloc(&d_d,  calc_per_loop * n_sys * sizeof(double)));
    gpuErrchk(hipMalloc(&d_ud, calc_per_loop * n_sys * sizeof(double)));
    gpuErrchk(hipMalloc(&d_x, calc_per_loop * n_sys  * sizeof(double)));   

}
/********//********//********//********//********//********//********//********//********//********/
void FokkerPlanckOnGPU::destroy(int device)
{
    hipSetDevice(device);
    hipFree(d_ld);hipFree(d_ud);hipFree(d_d);hipFree(d_x);
}
/********//********//********//********//********//********//********//********//********//********/
void FokkerPlanckOnGPU::SolveTridiagonal(double *ld, double *dd, double *ud, double *fin, int device)
{
    hipSetDevice(device);
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

    const int N     =  n_sys*calc_per_loop;        // --- Size of the linear system

    gpuErrchk(hipMemcpy(d_ld, ld, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 11 \n";
    gpuErrchk(hipMemcpy(d_d,  dd, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 12 \n";
    gpuErrchk(hipMemcpy(d_ud, ud, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 13 \n";

    // --- Allocating and defining dense device data vectors
    gpuErrchk(hipMemcpy(d_x, fin, N * sizeof(double), hipMemcpyHostToDevice));//std::cout << "\n 14 \n";

    // --- Solve for solution
    cusparseSafeCall(cusparseDgtsvStridedBatch(handle, calc_per_loop, d_ld, d_d, d_ud, d_x, n_sys, calc_per_loop));

    // --- Copy back into host
    hipMemcpy(fin, d_x, N * sizeof(double), hipMemcpyDeviceToHost);

    cusparseSafeCall(hipsparseDestroy(handle));
}

/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
