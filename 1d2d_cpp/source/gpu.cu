#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "gpu.h"

#define BLOCKSIZE_x 32
#define BLOCKSIZE_y 4

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) { exit(code); }
   }
}

extern "C" void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
    switch (error)
    {

        case HIPSPARSE_STATUS_SUCCESS:
            return "HIPSPARSE_STATUS_SUCCESS";

        case HIPSPARSE_STATUS_NOT_INITIALIZED:
            return "HIPSPARSE_STATUS_NOT_INITIALIZED";

        case HIPSPARSE_STATUS_ALLOC_FAILED:
            return "HIPSPARSE_STATUS_ALLOC_FAILED";

        case HIPSPARSE_STATUS_INVALID_VALUE:
            return "HIPSPARSE_STATUS_INVALID_VALUE";

        case HIPSPARSE_STATUS_ARCH_MISMATCH:
            return "HIPSPARSE_STATUS_ARCH_MISMATCH";

        case HIPSPARSE_STATUS_MAPPING_ERROR:
            return "HIPSPARSE_STATUS_MAPPING_ERROR";

        case HIPSPARSE_STATUS_EXECUTION_FAILED:
            return "HIPSPARSE_STATUS_EXECUTION_FAILED";

        case HIPSPARSE_STATUS_INTERNAL_ERROR:
            return "HIPSPARSE_STATUS_INTERNAL_ERROR";

        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
            return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

        case HIPSPARSE_STATUS_ZERO_PIVOT:
            return "HIPSPARSE_STATUS_ZERO_PIVOT";
    }

    return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
    if(HIPSPARSE_STATUS_SUCCESS != err) {
        fprintf(stderr, "CUSPARSE error in file '%s', line %Ndims \n objs %s\nerror %Ndims: %s\nterminating! \n objs",__FILE__, __LINE__,err, \
                                _cusparseGetErrorEnum(err)); \
        hipDeviceReset(); assert(0); \
    }
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
// -------------
// column (nump) derivative
// -------------
/********//********//********//********//********//********//********//********//********//********/
__global__ void e_times_derivative_p(double *f, double *df, int nump, int offset)
{  
    __shared__ double s_f[4][BLOCKSIZE_x + 4]; // 2-wide halo

    int ip   = threadIdx.x;
    int ix   = blockIdx.x*blockDim.y + threadIdx.y;

    int si = ip + 2;       // local i for shared memory access + halo offset
    int sj = threadIdx.y; // local j for shared memory access

    int globalIdx = offset + ix * nump + ip;

    s_f[sj][si] = f[globalIdx];

    __syncthreads();

    // fill in periodic images in shared memory array 
    if (ix < 2) 
    {
        s_f[sj][si-2]  = s_f[sj][si+BLOCKSIZE_x-3];
        s_f[sj][si+BLOCKSIZE_x] = s_f[sj][si+1];   
    }

    __syncthreads();

    df[globalIdx] = 
                        ( (4./3.) * ( s_f[sj][si+1] - s_f[sj][si-1] )
                        - (1./6.) * ( s_f[sj][si+2] - s_f[sj][si-2] ) );
}
/********//********//********//********//********//********//********//********//********//********/
// -------------
// column (numx) derivative
// -------------
/********//********//********//********//********//********//********//********//********//********/
/*
__global__ void derivative_x(double *f, double *df)
{
  __shared__ double s_f[BLOCKSIZE_y+4][4];

  int ip  = blockIdx.x*blockDim.x + threadIdx.x;
  int ix  = threadIdx.y;
  
  int si = threadIdx.x;
  int sj = ix + 2;

  int globalIdx =  ix * BLOCKSIZE_x + ip;

  s_f[sj][si] = f[globalIdx];

  __syncthreads();

  if (j < 2) {
    s_f[sj-2][si]  = s_f[sj+BLOCKSIZE_y-3][si];
    s_f[sj+BLOCKSIZE_y][si] = s_f[sj+1][si];
  }

  __syncthreads();

  df[globalIdx] = 
    ( 4./3. * ( s_f[sj+1][si] - s_f[sj-1][si] )
    - 1./6. * ( s_f[sj+2][si] - s_f[sj-2][si] ) );
} */
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
__global__ void v_times_derivative_x(double *f, double *df, int numx, int offset)
{
    __shared__ double s_f[BLOCKSIZE_y+4][32];

    int ip  = blockIdx.x*blockDim.x + threadIdx.x;
    int si = threadIdx.x;

    for (int ix = threadIdx.y; ix < BLOCKSIZE_y; ix += blockDim.y) {
    int globalIdx = offset + ix * BLOCKSIZE_x + ip;
    int sj = ix + 2;
    s_f[sj][si] = f[globalIdx];
    }

    __syncthreads();

    int sj = threadIdx.y + 2;
    if (sj < 2) 
    {
        s_f[sj-2][si]  = s_f[sj+BLOCKSIZE_y-3][si];
        s_f[sj+BLOCKSIZE_y][si] = s_f[sj+1][si];   
    }

    __syncthreads();

    for (int ix = threadIdx.y; ix < BLOCKSIZE_y; ix += blockDim.y) 
    {
        int globalIdx = offset + ix * BLOCKSIZE_x + ip;
        int sj = ix + 2;
        df[globalIdx] = ( (4./3.) * ( s_f[sj+1][si] - s_f[sj-1][si] )
                            - (1./6.) * ( s_f[sj+2][si] - s_f[sj-2][si] ) );
    }
}
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
void GPU_interface_routines::setupTDsolve(double *d_ld, double *d_d, double *d_ud, double *d_x, int device)
{
    hipSetDevice(device);

    gpuErrchk(hipMalloc(&d_ld, N * sizeof(double)));
    gpuErrchk(hipMalloc(&d_d,  N * sizeof(double)));
    gpuErrchk(hipMalloc(&d_ud, N * sizeof(double)));
}


void GPU_interface_routines::TDsolve( int calculations_per_loop, int n_systems,
                            double *ld, 
                            double *dd, 
                                  double *ud,      
                                  double *fin,// int device)
                                  double *d_ld, double *d_d, double *d_ud, double *d_x)
{
    hipSetDevice(device);
    // --- Initialize cuSPARSE
    hipsparseHandle_t handle;    cusparseSafeCall(hipsparseCreate(&handle));

    const int N     =  n_systems*calculations_per_loop;        // --- Size of the linear system

    // // --- Lower diagonal, diagonal and upper diagonal of the system matrix
    // double *d_ld;   gpuErrchk(hipMalloc(&d_ld, N * sizeof(double)));
    // double *d_d;    gpuErrchk(hipMalloc(&d_d,  N * sizeof(double)));
    // double *d_ud;   gpuErrchk(hipMalloc(&d_ud, N * sizeof(double)));

    gpuErrchk(hipMemcpy(d_ld, ld, N * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_d,  dd, N * sizeof(double), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_ud, ud, N * sizeof(double), hipMemcpyHostToDevice));

    // --- Allocating and defining dense device data vectors
    // double *d_x;        gpuErrchk(hipMalloc(&d_x, N * sizeof(double)));   
    gpuErrchk(hipMemcpy(d_x, fin, N * sizeof(double), hipMemcpyHostToDevice));

    // --- Solve for solution
    cusparseSafeCall(cusparseDgtsvStridedBatch(handle, calculations_per_loop, d_ld, d_d, d_ud, d_x, n_systems, calculations_per_loop));

    // --- Copy back into host
    hipMemcpy(fin, d_x, N * sizeof(double), hipMemcpyDeviceToHost);

    // hipFree(d_ld);hipFree(d_ud);hipFree(d_d);hipFree(d_x);
    cusparseSafeCall(hipsparseDestroy(handle));
}
void GPU_interface_routines::setupTDsolve(double *d_ld, double *d_d, double *d_ud, double *d_x)
{
    hipFree(d_ld);hipFree(d_ud);hipFree(d_d);hipFree(d_x);
}

/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
void GPU_interface_routines::calc_fieldxdf(int numx, int nump, int numdist, double *fin, 
                                            // double *dxf, double *dvf, 
                                            // double *ex, double *vtemp, 
                                            int device)
{
    hipSetDevice(device);

    /// ------------------------------
    /// Row and Column Sizing
    /// ------------------------------
    // const int nump = _nump;
    // const int numx = _numx;
    // const int numdist = _numdist;
    const int totalsize = (2*numdist)*numx*nump;
    
    /// ------------------------------
    /// Allocate and initialize Arrays
    /// ------------------------------
    double *d_dxf;     gpuErrchk(hipMalloc(&d_dxf,    totalsize  * sizeof(double)));
    gpuErrchk(hipMemset(d_dxf, 0.,             totalsize  * sizeof(double)));

    double *d_dvf;     gpuErrchk(hipMalloc(&d_dvf,    totalsize  * sizeof(double)));
    gpuErrchk(hipMemset(d_dvf, 0.,             totalsize  * sizeof(double)));

    // double *d_vtemp;   gpuErrchk(hipMalloc(&d_vtemp,  nump    * sizeof(double)));
    // gpuErrchk(hipMemcpy(d_vtemp, vtemp,    nump  * sizeof(double), hipMemcpyHostToDevice));

    // double *d_ex;      gpuErrchk(hipMalloc(&d_ex,     numx    * sizeof(double)));
    // gpuErrchk(hipMemcpy(d_ex,  ex,          numx  * sizeof(double), hipMemcpyHostToDevice));

    double *d_fin;        gpuErrchk(hipMalloc(&d_fin, totalsize * sizeof(double)));   
    gpuErrchk(hipMemcpy(d_fin, fin, totalsize * sizeof(double), hipMemcpyHostToDevice));
    
    /// ------------------------------
    /// Parallelization Grid on GPU
    /// ------------------------------
    dim3 threadsPerBlock(BLOCKSIZE_x, BLOCKSIZE_y);
    dim3 numBlocks(nump / threadsPerBlock.x, numx / threadsPerBlock.y);

    /// ------------------------------
    /// vgradf or Edfdv
    /// ------------------------------
    // e_times_derivative_p<<<numBlocks, threadsPerBlock>>>(d_fin, d_ex,       d_dvf, nump);
    int baseidx;
    for (int id(0); id < 2*numdist; ++id)
    {
        baseidx = id*numx*nump;
        
        // gpuErrchk(hipMemcpy(d_dxftemp, d_fin+baseidx, nump*numx * sizeof(double), hipMemcpyDeviceToDevice));
        v_times_derivative_x<<<numBlocks, threadsPerBlock>>>(d_fin, d_dxf, nump, baseidx);
        // gpuErrchk(hipMemcpy(d_fin+baseidx, d_dxf, nump*numx * sizeof(double), hipMemcpyDeviceToDevice));

    }
    
    /// ------------------------------
    /// Back to CPU
    /// ------------------------------
    // thrust::copy(d_dvf,d_dvf+totalsize,dvf.begin());
    gpuErrchk(hipMemcpy(fin, d_dxf, totalsize * sizeof(double), hipMemcpyDeviceToHost));


    /// ------------------------------
    /// Free resources
    /// ------------------------------
    hipFree(d_fin);hipFree(d_dxf);hipFree(d_dvf);
    // hipFree(d_ex);hipFree(d_vtemp);

}

/********//********//********//********//********//********//********//********//********//********/
/********//********//********//********//********//********//********//********//********//********/
